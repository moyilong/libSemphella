#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "main_use.h"

__global__ void __CUDA_kernel_xor_crypt(char *data, int password_len, char *password)
{

}


char __CUDA_xor_crypt(string password, char *data, int len)
{
	char *pas;
	char *dat_h;
	hipMalloc((void**)pas, password.size());
	hipMalloc((void**)dat_h, len);
	hipMemcpy(pas, password.data(),password.size(), hipMemcpyHostToDevice);
	hipMemcpy(dat_h, data,password.size(), hipMemcpyHostToDevice);
	__CUDA_kernel_xor_crypt << <len, 1 >> >(dat_h, password.size(), pas);
	hipDeviceSynchronize();
	hipMemcpy(data, dat_h, len, hipMemcpyDeviceToHost);
}